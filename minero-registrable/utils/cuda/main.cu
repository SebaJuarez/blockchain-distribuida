// main.cu
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <string> // Para std::string y std::to_string
#include <hip/hip_runtime.h>
#include "md5.cuh" // Asegúrate de que esta cabecera esté disponible

using byte = unsigned char;
using word = unsigned int; // Asegúrate de que 'word' sea unsigned int aquí también

// Rango de nonces a probar en cada batch (≤ 65 535)
static constexpr int BATCH_SIZE = 1 << 15;  // 32 768 nonces por llamada

int main(int argc, char** argv) {
    if (argc != 5) {
        std::fprintf(stderr, "Uso: %s <prefijo_hex> <cadena> <inicio> <fin>\n", argv[0]);
        return EXIT_FAILURE;
    }

    const char* prefix = argv[1];
    const std::string base = argv[2]; // Usar std::string para facilitar la concatenación
    int prefix_len = std::strlen(prefix);
    int base_len = base.length();

    // Leer rango de búsqueda
    unsigned long inicio = std::strtoul(argv[3], nullptr, 10);
    unsigned long fin = std::strtoul(argv[4], nullptr, 10);
    if (fin < inicio) {
        std::fprintf(stderr, "Error: fin debe ser mayor o igual a inicio\n");
        return EXIT_FAILURE;
    }

    // Máxima longitud posible para una entrada (longitud máxima de nonce string + longitud de base)
    // Un unsigned long puede tener hasta 20 dígitos (para 18.4 quintillones)
    // Se inicializa después de que base_len tiene un valor
    // No es constexpr porque base_len no lo es.
    const int MAX_INPUT_LEN = 20 + base_len; // Sin el +1 ya que std::string::length() no incluye el nulo, y memcpy usa esa longitud.

    // buffers host/device
    byte* h_in_data = nullptr;
    word* h_in_lengths = nullptr; // Nuevo buffer para almacenar las longitudes
    byte* h_out = nullptr;

    hipHostAlloc(&h_in_data, (size_t)BATCH_SIZE * MAX_INPUT_LEN, hipHostMallocDefault); // Cast a size_t
    hipHostAlloc(&h_in_lengths, (size_t)BATCH_SIZE * sizeof(word), hipHostMallocDefault); // Asignar para las longitudes, cast a size_t
    hipHostAlloc(&h_out, (size_t)BATCH_SIZE * 16, hipHostMallocDefault); // Cast a size_t

    unsigned long global_start = inicio;
    bool done = false;
    unsigned long found_nonce = (unsigned long)(-1);
    byte found_hash[16];

    while (!done && global_start <= fin) {
        // Ajustar el tamaño del batch si nos acercamos al fin del rango
        int this_batch = BATCH_SIZE;
        if (global_start + this_batch - 1 > fin) {
            this_batch = (int)(fin - global_start + 1);
        }

        // Preparar input batch
        for (int i = 0; i < this_batch; ++i) {
            unsigned long nonce = global_start + i;
            std::string nonce_str = std::to_string(nonce);
            std::string full_input_str = nonce_str + base;

            // Copiar la cadena al buffer de entrada y almacenar su longitud
            std::memcpy(h_in_data + i * MAX_INPUT_LEN, full_input_str.c_str(), full_input_str.length());
            h_in_lengths[i] = (word)full_input_str.length(); // Cast a 'word' (unsigned int)
        }

        // Llamar batch kernel
        mcm_cuda_md5_hash_batch(
            h_in_data,
            h_in_lengths, // Pasar el buffer de longitudes
            (word)MAX_INPUT_LEN, // Pasar la máxima longitud para el cálculo de offset (cast a word)
            h_out,
            (word)this_batch); // Cast a word

        // Buscar solución en resultados
        for (int i = 0; i < this_batch; ++i) {
            bool ok = true;
            for (int b = 0; b < prefix_len; ++b) {
                byte v = h_out[i * 16 + (b / 2)];
                char c = (b % 2 == 0)
                    ? "0123456789abcdef"[(v >> 4) & 0xF]
                    : "0123456789abcdef"[v & 0xF];
                if (c != prefix[b]) {
                    ok = false;
                    break;
                }
            }
            if (ok) {
                done = true;
                found_nonce = global_start + i;
                memcpy(found_hash, h_out + i * 16, 16);
                break;
            }
        }

        global_start += this_batch;
    }

    if (done) {
        // imprimir resultado
        char hex[33] = {0};
        static const char* digs = "0123456789abcdef";
        for (int i = 0; i < 16; ++i) {
            hex[2 * i]      = digs[(found_hash[i] >> 4) & 0xF];
            hex[2 * i + 1] = digs[found_hash[i] & 0xF];
        }
        std::printf("Nonce encontrado: %lu\n", found_nonce);
        std::printf("Hash resultante: %s\n", hex);
    } else {
        std::printf("No se encontro un hash que comience con '%s' en el rango [%lu, %lu]\n",
                     prefix, inicio, fin);
    }

    // limpiar
    hipHostFree(h_in_data);
    hipHostFree(h_in_lengths);
    hipHostFree(h_out);

    return EXIT_SUCCESS;
}