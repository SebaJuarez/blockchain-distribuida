// main.cu
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include "md5.cuh"

using byte = unsigned char;
using word = unsigned short;

// Rango de nonces a probar en cada batch (≤ 65 535)
static constexpr int BATCH_SIZE = 1 << 15;  // 32 768 nonces por llamada

int main(int argc, char** argv) {
    if (argc != 5) {
        std::fprintf(stderr, "Uso: %s <prefijo_hex> <cadena> <inicio> <fin>\n", argv[0]);
        return EXIT_FAILURE;
    }

    const char* prefix = argv[1];
    const char* base   = argv[2];
    int prefix_len = std::strlen(prefix);
    int base_len   = std::strlen(base);

    // Leer rango de búsqueda
    unsigned long inicio = std::strtoul(argv[3], nullptr, 10);
    unsigned long fin    = std::strtoul(argv[4], nullptr, 10);
    if (fin < inicio) {
        std::fprintf(stderr, "Error: fin debe ser mayor o igual a inicio\n");
        return EXIT_FAILURE;
    }

    // buffers host/device
    byte*  h_in   = nullptr;
    byte*  h_out  = nullptr;
    bool*   d_found = nullptr;
    int*    d_found_idx = nullptr;

    hipHostAlloc(&h_in,    BATCH_SIZE * (base_len + 16), hipHostMallocDefault);
    hipHostAlloc(&h_out,   BATCH_SIZE * 16,             hipHostMallocDefault);
    hipMalloc   (&d_found, sizeof(bool));
    hipMalloc   (&d_found_idx, sizeof(int));
    hipMemset   (d_found, 0, sizeof(bool));

    unsigned long global_start = inicio;
    bool done = false;
    unsigned long found_nonce = (unsigned long)(-1);
    byte found_hash[16];

    while (!done && global_start <= fin) {
        // Ajustar el tamaño del batch si nos acercamos al fin del rango
        int this_batch = BATCH_SIZE;
        if (global_start + this_batch - 1 > fin) {
            this_batch = (int)(fin - global_start + 1);
        }

        // Preparar input batch
        for (int i = 0; i < this_batch; ++i) {
            unsigned long nonce = global_start + i;
            int len = std::sprintf(reinterpret_cast<char*>(h_in + i * (base_len + 16)),
                                   "%lu", nonce);
            std::memcpy(h_in + i * (base_len + 16) + len,
                        base, base_len);
            std::memset(h_in + i * (base_len + 16) + len + base_len,
                        0, 16 - len);
        }

        // Reset flag y llamar batch kernel
        hipMemset(d_found, 0, sizeof(bool));
        mcm_cuda_md5_hash_batch(
          h_in,
          (word)(base_len + 16),
          h_out,
          (word)this_batch);

        // Buscar solución en resultados
        for (int i = 0; i < this_batch; ++i) {
            bool ok = true;
            for (int b = 0; b < prefix_len; ++b) {
                byte v = h_out[i * 16 + (b / 2)];
                char c = (b % 2 == 0)
                  ? "0123456789abcdef"[(v >> 4) & 0xF]
                  : "0123456789abcdef"[v & 0xF];
                if (c != prefix[b]) {
                    ok = false;
                    break;
                }
            }
            if (ok) {
                done = true;
                found_nonce = global_start + i;
                memcpy(found_hash, h_out + i * 16, 16);
                break;
            }
        }

        global_start += this_batch;
    }

    if (done) {
        // imprimir resultado
        char hex[33] = {0};
        static const char* digs = "0123456789abcdef";
        for (int i = 0; i < 16; ++i) {
            hex[2 * i]     = digs[(found_hash[i] >> 4) & 0xF];
            hex[2 * i + 1] = digs[found_hash[i] & 0xF];
        }
        std::printf("Nonce encontrado: %lu\n", found_nonce);
        std::printf("Hash resultante: %s\n", hex);
    } else {
        std::printf("No se encontro un hash que comience con '%s' en el rango [%lu, %lu]\n",
                    prefix, inicio, fin);
    }

    // limpiar
    hipHostFree(h_in);
    hipHostFree(h_out);
    hipFree(d_found);
    hipFree(d_found_idx);

    return EXIT_SUCCESS;
}