#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "md5.cuh"

using byte = unsigned char;
using word = unsigned short;

int main(int argc, char** argv) {
    if (argc < 2) {
        std::fprintf(stderr, "Uso: %s <texto1> [texto2 ... textoN]\n", argv[0]);
        return EXIT_FAILURE;
    }

    int n = argc - 1;
    std::vector<const char*> inputs(n);
    for (int i = 0; i < n; ++i) inputs[i] = argv[i+1];

    // 1) Determinar la longitud máxima
    size_t maxlen = 0;
    for (auto s : inputs) maxlen = std::max(maxlen, std::strlen(s));

    // 2) Crear buffer de entrada: n mensajes de longitud `maxlen`, padded con 0
    byte* h_in = (byte*)malloc(maxlen * n);
    for (int i = 0; i < n; ++i) {
        size_t L = std::strlen(inputs[i]);
        std::memcpy(h_in + i*maxlen, inputs[i], L);
        if (L < maxlen) std::memset(h_in + i*maxlen + L, 0, maxlen - L);
    }

    // 3) Reservar salida host (16 bytes de digest por mensaje)
    byte* h_out = (byte*)malloc(16 * n);

    // 4) Llamar a la función batch (gestiona hipMalloc, kernel y hipMemcpy)
    //    Parámetros: (in, length de CADA mensaje, out, cuántos mensajes)
    mcm_cuda_md5_hash_batch(h_in, (word)maxlen, h_out, (word)n);

    // 5) Imprimir resultados
    for (int i = 0; i < n; ++i) {
        std::printf("MD5(\"%s\") = ", inputs[i]);
        byte* digest = h_out + 16*i;
        for (int b = 0; b < 16; ++b) {
            std::printf("%02x", digest[b]);
        }
        std::printf("\n");
    }

    // 6) Liberar
    free(h_in);
    free(h_out);
    return EXIT_SUCCESS;
}