// main.cu
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include "md5.cuh"

using byte = unsigned char;
using word = unsigned short;

// Rango de nonces a probar en cada batch (≤ 65 535)
static constexpr int BATCH_SIZE = 1 << 15;  // 32 768 nonces por llamada

int main(int argc, char** argv) {
    if (argc != 3) {
        std::fprintf(stderr, "Uso: %s <prefijo_hex> <cadena>\n", argv[0]);
        return EXIT_FAILURE;
    }

    const char* prefix = argv[1];
    const char* base   = argv[2];
    int prefix_len = std::strlen(prefix);
    int base_len   = std::strlen(base);

    // buffers host/device
    // h_in: para enviar BATCH_SIZE concatenaciones de (nonce + base), todas con igual longitud
    // h_out: para recuperar BATCH_SIZE hashes (16 bytes c/u)
    // d_found: flag para indicar cuándo se encontró
    byte*  h_in   = nullptr;
    byte*  h_out  = nullptr;
    bool   *d_found = nullptr;
    int    *d_found_idx = nullptr;

    hipHostAlloc(&h_in,    BATCH_SIZE * (base_len + 16), hipHostMallocDefault);
    hipHostAlloc(&h_out,   BATCH_SIZE * 16,             hipHostMallocDefault);
    hipMalloc   (&d_found, sizeof(bool));
    hipMalloc   (&d_found_idx, sizeof(int));
    hipMemset   (d_found, 0, sizeof(bool));

    int  global_start = 0;
    bool done = false;
    int  found_nonce = -1;
    byte found_hash[16];

    while (!done) {
        // 1) Prepárate BATCH_SIZE nonces [global_start .. global_start + BATCH_SIZE)
        int this_batch = BATCH_SIZE;
        // (no ajustamos si sobrepasa, simplificamos; podría adaptarse)
        for (int i = 0; i < this_batch; ++i) {
            int nonce = global_start + i;
            // escribe nonce como string
            int len = std::sprintf(reinterpret_cast<char*>(h_in + i*(base_len+16)),
                                   "%d", nonce);
            // rellena con la base
            std::memcpy(h_in + i*(base_len+16) + len,
                        base, base_len);
            // ceros al final
            std::memset(h_in + i*(base_len+16) + len + base_len,
                        0, 16 - len);
        }

        // 2) copia los datos y el flag a device
        hipMemset(d_found, 0, sizeof(bool));
        hipMemcpyToSymbol(HIP_SYMBOL(/*mcm_cuda_md5_input*/ nullptr), nullptr, 0); // no-op
        // Llamada batch: internamente ejecuta un kernel de BATCH_SIZE hilos
        mcm_cuda_md5_hash_batch(
          h_in,
          (word)(base_len + 16),   // longitud fija de cada mensaje (_nonce_str + base)
          h_out,
          (word)this_batch);

        // 3) después de la llamada, escanea en h_out si alguno cumple el prefijo
        for (int i = 0; i < this_batch; ++i) {
            // convierte los primeros prefix_len nibbles a chars
            bool ok = true;
            for (int b = 0; b < prefix_len; ++b) {
                // cada byte de hash → 2 hex chars
                byte v = h_out[i*16 + (b/2)];
                char c = (b%2==0)
                  ? "0123456789abcdef"[(v>>4)&0xF]
                  : "0123456789abcdef"[v&0xF];
                if (c != prefix[b]) { ok = false; break; }
            }
            if (ok) {
                done = true;
                found_nonce = global_start + i;
                memcpy(found_hash, h_out + i*16, 16);
                break;
            }
        }

        global_start += this_batch;
    }

    // Imprime resultado
    // hash en hex:
    char hex[33] = {0};
    static const char* digs = "0123456789abcdef";
    for (int i = 0; i < 16; ++i) {
        hex[2*i  ] = digs[(found_hash[i] >> 4) & 0xF];
        hex[2*i+1] = digs[found_hash[i] & 0xF];
    }
    std::printf("Nonce encontrado: %d\n", found_nonce);
    std::printf("Hash resultante: %s\n", hex);

    // limpia
    hipHostFree(h_in);
    hipHostFree(h_out);
    hipFree(d_found);
    hipFree(d_found_idx);
    return EXIT_SUCCESS;
}