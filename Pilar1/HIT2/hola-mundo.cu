
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void holaMundo() {
    printf("Hola Mundo desde GPU!\n");
}

int main() {
    holaMundo<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
